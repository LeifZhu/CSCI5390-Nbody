#include "hip/hip_runtime.h"
#include "main.h"
#include <cstdlib>
#include <cstring>
#include <cstdio>
#include <cmath>

#include "NBody.cuh"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cassert>


double getRandom(double min, double max)
{
	double r = (double)rand() / RAND_MAX;
	return r*(max - min) + min;
}

__global__ void addTrail(unsigned char* d_buffer,
        const double decay,
        const unsigned int screen_width,
        const unsigned int screen_height)

{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    if(x >= screen_width || y >= screen_height) return;
    d_buffer[x * screen_height * 3 + y * 3 + 0] *= decay;
    d_buffer[x * screen_height * 3 + y * 3 + 1] *= decay;
    d_buffer[x * screen_height * 3 + y * 3 + 2] *= decay;
}

__global__ void updateFrame(struct body *d_bodies,
        unsigned char* d_buffer,
        unsigned int screen_width,
        unsigned int screen_height,
        unsigned int num_bodies)
{
    unsigned int i =  blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= num_bodies) return;

    unsigned int x = (unsigned int)((d_bodies[i].x + 1.0)/2.0 * screen_width);
    unsigned int y = (unsigned int)((d_bodies[i].y + 1.0)/2.0 * screen_height);
    d_buffer[x * screen_height * 3 + y * 3 + 0] = 255;
    d_buffer[x * screen_height * 3 + y * 3 + 1] = 255;
    d_buffer[x * screen_height * 3 + y * 3 + 2] = 255;
}

void rasterize(struct body* d_bodies, unsigned char* d_buffer, unsigned char* h_buffer)
{
	/**
	rasterize the bodies from x,y: (-1,-1) to (1,1) according to some kind of formula

	Note: You can change the code for better visualization
	As the following code can be parallelized, you can optimize this routine with CUDA.

	\param bodies A collection of bodies (located on the device).
	\param buffer the RGB buffer for screen display (located on the host).
	*/
    unsigned int bufferSize = SCREEN_WIDTH * SCREEN_HEIGHT * 3 * sizeof(unsigned char);

    // add trail effect
    dim3 gridShape(1, SCREEN_HEIGHT);
    dim3 blockShape((SCREEN_WIDTH + gridShape.x - 1) / gridShape.x,
                    (SCREEN_HEIGHT + gridShape.y - 1) / gridShape.y);
    addTrail<<<gridShape, blockShape>>>(d_buffer, 0.98, SCREEN_WIDTH, SCREEN_HEIGHT);

    // update frame
    unsigned int groupSize = 8;
    updateFrame<<<groupSize, (NUM_BODIES + groupSize - 1) / groupSize>>>(
            d_bodies, d_buffer, SCREEN_WIDTH, SCREEN_HEIGHT, NUM_BODIES);

    hipMemcpy(h_buffer, d_buffer, bufferSize, hipMemcpyDeviceToHost);
}

void initializeNBodyCuda(struct body * &d_bodies, unsigned char* &d_buffer)
{
	/**
	initialize the bodies, then copy to the CUDA device memory
	return the device pointer so that it can be reused in the NBodyTimestepCuda function.
	*/
	// initialize the position and velocity
	// you can implement own initial conditions to form a sprial/ellipse galaxy, have fun.
	struct body* h_bodies = new struct body[NUM_BODIES];
	for(int i = 0; i < NUM_BODIES; i++)
    {
		h_bodies[i].x = getRandom(-1.0, 1.0);
        h_bodies[i].y = getRandom(-1.0, 1.0);
        h_bodies[i].vx = getRandom(-1.0, 1.0);
        h_bodies[i].vy = getRandom(-1.0, 1.0);
        //bodies[i].vx = 0;
        //bodies[i].vy = 0;
        h_bodies[i].m = getRandom(1e5, 1e7);
    }
    //d_body points to device memory
	hipMalloc((void**)&d_bodies, NUM_BODIES * sizeof(struct body));
	hipMemcpy(d_bodies, h_bodies, NUM_BODIES * sizeof(struct body), hipMemcpyHostToDevice);
	// after copy initialized bodies to device, we can recycle host memory now
	delete [] h_bodies;

	// initialize buffer
    unsigned int bufferSize = SCREEN_WIDTH * SCREEN_HEIGHT * 3 * sizeof(unsigned char);
    hipMalloc((void**)&d_buffer, bufferSize);
    hipMemset(d_buffer, 0, bufferSize);
}

void freeMem(struct body* d_bodies, unsigned char* d_buffer)
{
    // if bodies is located in host memory, use delete
    // if in device memory, use hipFree
    hipFree(d_buffer);
    hipFree(d_bodies);
}


__global__ void updateBodies(struct body* d_bodies,
        double dt,
        double rx,
        double ry,
        bool cursor)
{
    __shared__ double ax[NUM_BODIES], ay[NUM_BODIES];
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    // initialization
    ax[tid] = 0;
    ay[tid] = 0;
    __syncthreads();


    if(tid != bid) 
	{
        double dx = d_bodies[tid].x - d_bodies[bid].x;
        double dy = d_bodies[tid].y - d_bodies[bid].y;
        double dist3 = pow(dx * dx + dy * dy, 1.5) + eps;
        ax[tid] = G * d_bodies[tid].m / dist3 * dx;
        ay[tid] = G * d_bodies[tid].m / dist3 * dy;
	}
	__syncthreads();
    /* TODO: here we assume NUM_BODIES is 2^k for some k,
      or else we need extra judgement for reduction */
	for (unsigned int s = NUM_BODIES >> 1; s > 0; s >>= 1)
	{
		if (tid < s)
		{
			ax[tid] += ax[tid + s];
			ay[tid] += ay[tid + s];
		}
		__syncthreads();
	}
    
    // TODO: try seperate the following code to another kernel
    // because in remaining part, there is only one thread alive on blocks
    if(tid == 0)
    {
        if(cursor)
        {
            double dx = rx - d_bodies[bid].x;
            double dy = ry - d_bodies[bid].y;
            double dist3 = pow(dx * dx + dy * dy, 1.5) + eps;
            // magnify the effect of cursor
            ax[0] += 1e7 * G * cursor_weight / dist3 * dx;
            ay[0] += 1e7 * G * cursor_weight / dist3 * dy;
        }
        double vx = d_bodies[bid].vx + ax[0] * dt;
        double vy = d_bodies[bid].vy + ay[0] * dt;
        double x = d_bodies[bid].x + vx * dt;
        double y = d_bodies[bid].y + vy * dt;
        // after collsision, the actual positon should
        // be symmetric correspoding to boundary
        // here we assume the velocity won't be too large,
        // so at most one collision can happen between two frames
        if(x > 1.0)
        {
           vx *= -collision_damping;
           // more precicsely, x = 1.6 - 0.6*x;
           x = 2.0 - x;
        }
        else if(x < -1.0)
        {
            vx *= -collision_damping;
            x = -2.0 -x;
        }
        if(y > 1.0)
        {
            vy *= -collision_damping;
            y = 2.0 - y;
        }
        else if(y < -1.0)
        {
            vy *= -collision_damping;
            y = -2.0 -y;
        }
        d_bodies[bid].x = x;
        d_bodies[bid].y = y;
        d_bodies[bid].vx = vx;
        d_bodies[bid].vy = vy;
    }
}

void NBodyTimestepCuda(struct body* d_bodies, double rx, double ry, bool cursor)
{
    /**
    Compute a time step on the CUDA device.
    \param bodies A collection of bodies (located on the device).
    \param rx position x of the cursor.
    \param ry position y of the cursor.
    \param cursor Enable the mouse interaction if true (adding a weight = cursor_weight body in the computation).
    */
    // TODO: current version has low scalability, try to write new kernel to hadle more bodies
    assert(NUM_BODIES <= 1024);
    double dt = 1e-3;
    updateBodies<<<NUM_BODIES, NUM_BODIES>>>(d_bodies, dt, rx, ry, cursor);
}
